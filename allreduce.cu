/*
 * This file tests NCCL allreduce between 2 nodes.
 *
 * Copyright (c) 2024 by Yue Yu
 */
#include <nccl.h>

#include <hip/hip_runtime.h>

#include <hip/hip_fp16.h>

#include <cstdio>
#include <random>
#include <vector>
#include <thread>

constexpr size_t cnt = 1024 * 1024 * 1024 / sizeof(half);
constexpr int    dSize = 8;
constexpr float  low = 0.f,
                 high = 1.f;
constexpr int    seed = 595;

#define CUDACHECK(cmd) do { \
  hipError_t e = cmd;      \
  if (e != hipSuccess) {   \
    std::printf("CUDA error %s: %d '%s'\n", __FILE__,__LINE__, hipGetErrorString(e));\
    exit(EXIT_FAILURE);     \
  }                         \
} while(0)


#define NCCLCHECK(cmd) do { \
  ncclResult_t r = cmd;     \
  if (r != ncclSuccess) {   \
    std::printf("NCCL error %s: %d '%s'\n", __FILE__,__LINE__, ncclGetErrorString(r)); \
    exit(EXIT_FAILURE);     \
  }                         \
} while(0)

template<typename S, typename T>
void random_fill(S* v, T low, T high)
{
    std::mt19937 gen(seed);
    std::uniform_real_distribution<float> dis(static_cast<float>(low),
                                              static_cast<float>(high));
    for (size_t i = 0; i < cnt; ++i) {
        v[i] = static_cast<S>(dis(gen));
    }
}

void threadFn(int rank, ncclUniqueId* id)
{
    CUDACHECK(hipSetDevice(rank));

    hipStream_t stream;
    CUDACHECK(hipStreamCreate(&stream));

    ncclComm_t comm;
    NCCLCHECK(ncclCommInitRank(&comm, dSize, *id, rank));

    half* hIn  = new half[cnt]; random_fill(hIn, low, high);
    half* hOut = new half[cnt];

    half* dIn;
    half* dOut;
    CUDACHECK(hipMalloc(&dIn , cnt * sizeof(half)));
    CUDACHECK(hipMalloc(&dOut, cnt * sizeof(half)));

    CUDACHECK(hipMemcpyAsync(dIn, hIn, cnt * sizeof(half), hipMemcpyHostToDevice, stream));
    NCCLCHECK(ncclAllReduce(
        reinterpret_cast<const void*>(dIn),
        reinterpret_cast<void*>(dOut),
        cnt,
        ncclHalf,
        ncclSum,
        comm,
        stream
    ));
    CUDACHECK(hipMemcpyAsync(hOut, dOut, cnt * sizeof(half), hipMemcpyDeviceToHost, stream));

    NCCLCHECK(ncclCommDestroy(comm));
    delete[] hIn;
    delete[] hOut;
    CUDACHECK(hipFree(dIn));
    CUDACHECK(hipFree(dOut));
}

int main(int argc, char* argv[])
{
    ncclUniqueId id;
    NCCLCHECK(ncclGetUniqueId(&id));
    
    std::vector<std::thread> workers; workers.reserve(dSize);
    for (int i = 0; i < dSize; ++i) {
        workers.emplace_back(
            threadFn,
            i,
            &id
        );
    }

    for (auto&& t : workers) {
        t.join();
    }
}